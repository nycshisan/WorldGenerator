#include "hip/hip_runtime.h"
#include "cuda_common.h"

#include <cmath>

#include "hip/hip_runtime.h"
#include ""

__global__ void add(int n, float* y) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	y[index] = 3.0f;
}

bool CMCudaTest(void) {
	int N = 1 << 16;
	float *y;

	// Allocate Unified Memory �C accessible from CPU or GPU
	hipMallocManaged(&y, N * sizeof(float));

	int blockSize = 256;
	int numBlocks = (N + blockSize - 1) / blockSize;
	CUDA_KERNAL_CALL(add, numBlocks, blockSize)(N, y);

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	// Check for errors
	float maxError = 0.0f;
	for (int i = 0; i < N; i++)
		maxError = fmax(maxError, fabs(y[i] - 3.0f));

	// Free memory
	hipFree(y);

	return fabs(maxError) < 1e-5f;
}
