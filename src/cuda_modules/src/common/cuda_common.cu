#include "hip/hip_runtime.h"
#include "pch.h"

__global__ void add(int n, float* y) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	y[index] = 3.0f;
}

bool CMCudaTest(void) {
	int N = 1 << 16;
	float *y;

	// Allocate Unified Memory �C accessible from CPU or GPU
	hipMallocManaged(&y, N * sizeof(float));

	CUDA_KERNAL_CALL(add, N)(N, y);

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	// Check for errors
	float maxError = 0.0f;
	for (int i = 0; i < N; i++)
		maxError = fmax(maxError, fabs(y[i] - 3.0f));

	// Free memory
	hipFree(y);

	return fabs(maxError) < 1e-5f;
}
